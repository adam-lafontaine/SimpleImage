#include "hip/hip_runtime.h"
#include "simage_cuda.cpp"
#include "../cuda/cuda_def.cuh"


using RGBA = simage::RGBA;
using RGB = simage::RGB;
using HSV = simage::HSV;
using YUV = simage::YUV;


template <typename T>
using DeviceView2D = simage::DeviceView2D<T>;

template <typename T, size_t N>
using DeviceChannelView2D = simage::DeviceChannelView2D<T, N>;

using DeviceView1u16 = simage::DeviceView1u16;

using DeviceView4u16 = simage::DeviceView4u16;
using DeviceView3u16 = simage::DeviceView3u16;
using DeviceView2u16 = simage::DeviceView2u16;

using DeviceView = simage::DeviceView;
using DeviceViewGray = simage::DeviceViewGray;

using DeviceViewRGBAu16 = simage::DeviceView4u16;
using DeviceViewRGBu16 = simage::DeviceView3u16;
using DeviceViewHSVu16 = simage::DeviceView3u16;
using DeviceViewYUVu16 = simage::DeviceView3u16;


class ChannelXY
{
public:
	u32 ch;
	u32 x;
	u32 y;
};


class RGBu16
{
public:
    u16 red;
    u16 green;
    u16 blue;
};


class HSVu16
{
public:
    u16 hue;
    u16 sat;
    u16 val;
};

constexpr u8 CH_U8_MAX = 255;
constexpr u16 CH_U16_MAX = CH_U8_MAX * 256;


constexpr int THREADS_PER_BLOCK = 512;

constexpr int calc_thread_blocks(u32 n_threads)
{
    return (n_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}


/* conversion */

namespace gpuf
{
    template <typename T>
	GPU_CONSTEXPR_FUNCTION
	inline int id_cast(T channel)
	{
		return static_cast<int>(channel);
	}


    GPU_CONSTEXPR_FUNCTION
    inline r32 clamp(r32 value)
    {
        if (value < 0.0f)
        {
            value = 0.0f;
        }
        else if (value > 1.0f)
        {
            value = 1.0f;
        }

        return value;
    }


    GPU_CONSTEXPR_FUNCTION
    inline u32 round_to_u32(r32 value)
    {
        return (u32)(value + 0.5f);
    }


    GPU_CONSTEXPR_FUNCTION
    inline u16 round_to_u16(r32 value)
    {
        return (u16)(value + 0.5f);
    }


    GPU_CONSTEXPR_FUNCTION
    inline u8 round_to_u8(r32 value)
    {
        return (u8)(value + 0.5f);
    }


    template <typename T>
    GPU_CONSTEXPR_FUNCTION
    inline r32 channel_u8_to_r32(T value)
    {
        return (r32)value / CH_U8_MAX;
    }


    template <typename T>
    GPU_CONSTEXPR_FUNCTION
    inline r32 channel_u16_to_r32(T value)
    {
        return (r32)value / CH_U16_MAX;
    }
    
    
    GPU_CONSTEXPR_FUNCTION
    inline u8 channel_r32_to_u8(r32 value)
    {
        return gpuf::round_to_u8(gpuf::clamp(value) * CH_U8_MAX);
    }
    
    
    GPU_CONSTEXPR_FUNCTION
    inline u16 channel_r32_to_u16(r32 value)
    {
        return gpuf::round_to_u16(gpuf::clamp(value) * CH_U16_MAX);
    }


    template <typename T>
    GPU_CONSTEXPR_FUNCTION
    inline u16 channel_u8_to_u16(T value)
    {
        return (u16)(value * 256);
    }


    template <typename T>
    GPU_CONSTEXPR_FUNCTION
    inline u8 channel_u16_to_u8(T value)
    {
        return u8(value / 256);
    }


    template <typename T>
    GPU_CONSTEXPR_FUNCTION
    inline r32 to_grayscale_standard(T r, T g, T b)
    {
        constexpr r32 COEFF_R = 0.299f;
        constexpr r32 COEFF_G = 0.587f;
        constexpr r32 COEFF_B = 0.114f;

        return COEFF_R * r + COEFF_G * g + COEFF_B * b;
    }


    GPU_FUNCTION
    static HSVu16 rgb_u16_to_hsv_u16(u16 r, u16 g, u16 b)
    {
        auto max = (u16)umax(r, umax(g, b));
        auto min = (u16)umin(r, umin(g, b));

        u16 h = 0;
        u16 s = 0;
        u16 v = max;

        if (max == min)
        {
            return { h, s, v };
        }

        s = gpuf::channel_r32_to_u16((r32)(max - min) / max);

        auto const r_is_max = r == max;
        auto const r_is_min = r == min;
        auto const g_is_max = g == max;
        auto const g_is_min = g == min;
        auto const b_is_max = b == max;
        auto const b_is_min = b == min;

        constexpr u16 delta_h = CH_U16_MAX / 6;
        u16 delta_c = 0;        
        u16 h_id = 0;

        if (r_is_max && b_is_min)
        {
            h_id = 0;
            delta_c = g - min;
        }
        else if (g_is_max && b_is_min)
        {
            h_id = 1;
            delta_c = max - r;
        }
        else if (g_is_max && r_is_min)
        {
            h_id = 2;
            delta_c = b - min;
        }
        else if (b_is_max && r_is_min)
        {
            h_id = 3;
            delta_c = max - g;
        }
        else if (b_is_max && g_is_min)
        {
            h_id = 4;
            delta_c = r - min;
        }
        else
        {
            h_id = 5;
            delta_c = max - b;
        }

        h = (u16)(delta_h * (h_id + (r32)delta_c / (max - min)));

        return { h, s, v };
    }


    GPU_FUNCTION
    static RGBu16 hsv_u16_to_rgb_u16(u16 h, u16 s, u16 v)
    {
        if (v == 0 || s == 0)
        {
            return { v, v, v };
        }

        auto max = v;
        auto range = (r32)s / CH_U16_MAX * v;
        auto min = gpuf::round_to_u16(max - range);

        constexpr u16 delta_h = CH_U16_MAX / 6;

        auto d = (r32)h / delta_h;
        auto h_id = (int)d;
        auto ratio = d - h_id;

        auto rise = gpuf::round_to_u16(min + ratio * range);
        auto fall = gpuf::round_to_u16(max - ratio * range);

        u16 r = 0;
        u16 g = 0;
        u16 b = 0;

        switch (h_id)
        {
        case 0:
            r = max;
            g = rise;
            b = min;
            break;
        case 1:
            r = fall;
            g = max;
            b = min;
            break;
        case 2:
            r = min;
            g = max;
            b = rise;
            break;
        case 3:
            r = min;
            g = fall;
            b = max;
            break;
        case 4:
            r = rise;
            g = min;
            b = max;
            break;
        default:
            r = max;
            g = min;
            b = fall;
            break;
        }

        return { r, g, b };
    }


    GPU_FUNCTION
    static HSVu16 rgb_u8_to_hsv_u16(u8 r, u8 g, u8 b)
    {
        auto R = gpuf::channel_u8_to_u16(r);
        auto G = gpuf::channel_u8_to_u16(g);
        auto B = gpuf::channel_u8_to_u16(b);

        return gpuf::rgb_u16_to_hsv_u16(R, G, B);
    }


    GPU_FUNCTION
    static simage::RGBAu8 hsv_u16_to_rgba_u8(u16 h, u16 s, u16 v)
    {
        auto rgb = gpuf::hsv_u16_to_rgb_u16(h, s, v);

        return {
            gpuf::channel_u16_to_u8(rgb.red),
            gpuf::channel_u16_to_u8(rgb.green),
            gpuf::channel_u16_to_u8(rgb.blue),
            255
        };
    }

    
    GPU_CONSTEXPR_FUNCTION
    inline r32 rgb_to_yuv_y(r32 r, r32 g, r32 b)
    {
        constexpr r32 COEFF_R = 0.299f;
        constexpr r32 COEFF_G = 0.587f;
        constexpr r32 COEFF_B = 0.114f;

        return COEFF_R * r + COEFF_G * g + COEFF_B * b;
    }
    

    GPU_CONSTEXPR_FUNCTION
    inline r32 rgb_to_yuv_u(r32 r, r32 g, r32 b)
    {
        constexpr r32 COEFF_R = -0.14713f;
        constexpr r32 COEFF_G = -0.28886f;
        constexpr r32 COEFF_B = 0.436f;

        return COEFF_R * r + COEFF_G * g + COEFF_B * b + 0.5f;
    }
    

    GPU_CONSTEXPR_FUNCTION
    inline r32 rgb_to_yuv_v(r32 r, r32 g, r32 b)
    {
        constexpr r32 COEFF_R = 0.615f;
        constexpr r32 COEFF_G = -0.51499f;
        constexpr r32 COEFF_B = -0.10001f;

        return COEFF_R * r + COEFF_G * g + COEFF_B * b + 0.5f;
    }
    

    GPU_CONSTEXPR_FUNCTION
    inline r32 yuv_to_rgb_r(r32 y, r32 u, r32 v)
    {
        constexpr r32 COEFF_Y = 1.0f;
        //constexpr r32 COEFF_U = 0.0f;
        constexpr r32 COEFF_V = 1.13983f;

        //u -= 0.5f;
        v -= 0.5f;

        return COEFF_Y * y /*+ COEFF_U * u*/ + COEFF_V * v;
    }
    

    GPU_CONSTEXPR_FUNCTION
    inline r32 yuv_to_rgb_g(r32 y, r32 u, r32 v)
    {
        constexpr r32 COEFF_Y = 1.0f;
        constexpr r32 COEFF_U = -0.39465f;
        constexpr r32 COEFF_V = -0.5806f;

        u -= 0.5f;
        v -= 0.5f;

        return COEFF_Y * y + COEFF_U * u + COEFF_V * v;
    }
    

    GPU_CONSTEXPR_FUNCTION
    inline r32 yuv_to_rgb_b(r32 y, r32 u, r32 v)
    {
        constexpr r32 COEFF_Y = 1.0f;
        constexpr r32 COEFF_U = 2.03211f;
        //constexpr r32 COEFF_V = 0.0f;

        u -= 0.5f;
        //v -= 0.5f;

        return COEFF_Y * y + COEFF_U * u /*+ COEFF_V * v*/;
    }
}


/* row begin */

namespace gpuf
{
    template <typename T>
    GPU_FUNCTION
	inline T* row_begin(DeviceView2D<T> const& view, u32 y)
	{
		return view.matrix_data_ + (u64)((view.y_begin + y) * view.matrix_width + view.x_begin);
	}


    template <typename T, size_t N>
    GPU_FUNCTION
	inline T* channel_row_begin(DeviceChannelView2D<T, N> const& view, u32 y, u32 ch)
	{
		auto offset = (size_t)((view.y_begin + y) * view.channel_width_ + view.x_begin);

		return view.channel_data_[ch] + offset;
	}
}


/* xy_at */

namespace gpuf
{
    template <typename T>
    GPU_FUNCTION
	inline T* xy_at(DeviceView2D<T> const& view, u32 x, u32 y)
    {
        return gpuf::row_begin(view, y) + x;
    }


    template <typename T>
    GPU_FUNCTION
	inline T* xy_at(DeviceView2D<T> const& view, Point2Du32 const& pt)
    {
        return gpuf::row_begin(view, pt.y) + pt.x;
    }


    template <typename T, size_t N, typename CH>
    GPU_FUNCTION
    inline T* channel_xy_at(DeviceChannelView2D<T, N> const& view, u32 x, u32 y, CH ch)
    {
        return channel_row_begin(view, y, gpuf::id_cast(ch)) + x;
    }


    template <typename T, size_t N>
    GPU_FUNCTION
    inline T* channel_xy_at(DeviceChannelView2D<T, N> const& view, ChannelXY const& cxy)
    {
        return channel_row_begin(view, cxy.y, cxy.ch) + cxy.x;
    }
}


/* get_thread_xy */

namespace gpuf
{
    template <class VIEW>
	GPU_FUNCTION
	static Point2Du32 get_thread_xy(VIEW const& view, u32 thread_id)
	{
		// n_threads = width * height
		Point2Du32 p{};

		p.y = thread_id / view.width;
		p.x = thread_id - p.y * view.width;

		return p;
	}


    template <class VIEW>
	GPU_FUNCTION
	static ChannelXY get_thread_channel_xy(VIEW const& view, u32 thread_id)
	{
		auto width = view.width;
		auto height = view.height;

		ChannelXY cxy{};

		cxy.ch = thread_id / (width * height);
		cxy.y = (thread_id - width * height * cxy.ch) / width;
		cxy.x = (thread_id - width * height * cxy.ch) - cxy.y * width;

		return cxy;
	}
}


/* map rgb */

namespace gpu
{  
    GPU_KERNAL    
    static void rgba_u8_to_rgba_u16(DeviceView src, DeviceViewRGBAu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 4);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto rgba = gpuf::xy_at(src, cxy.x, cxy.y)->rgba;
        u8 s = 0;
        auto& d = *gpuf::channel_xy_at(dst, cxy);

        switch(cxy.ch)
        {
        case gpuf::id_cast(RGBA::R):
            s = rgba.red;
            break;
        case gpuf::id_cast(RGBA::G):
            s = rgba.green;
            break;
        case gpuf::id_cast(RGBA::B):
            s = rgba.blue;
            break;
        case gpuf::id_cast(RGBA::A):
            s = rgba.alpha;
            break;
        default:
            return;
        }

        d = gpuf::channel_u8_to_u16(s);
    }


    GPU_KERNAL    
    static void rgb_u8_to_rgb_u16(DeviceView src, DeviceViewRGBu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 3);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto rgba = gpuf::xy_at(src, cxy.x, cxy.y)->rgba;
        u8 s = 0;
        auto& d = *gpuf::channel_xy_at(dst, cxy);

        switch(cxy.ch)
        {
        case gpuf::id_cast(RGB::R):
            s = rgba.red;
            break;
        case gpuf::id_cast(RGB::G):
            s = rgba.green;
            break;
        case gpuf::id_cast(RGB::B):
            s = rgba.blue;
            break;
        default:
            return;
        }

        d = gpuf::channel_u8_to_u16(s);
    }
    

    GPU_KERNAL
    static void rgba_u16_to_rgba_u8(DeviceViewRGBAu16 src, DeviceView dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 4);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto s = gpuf::channel_u16_to_u8(*gpuf::channel_xy_at(src, cxy));
        auto& d = gpuf::xy_at(dst, cxy.x, cxy.y)->rgba;

        switch(cxy.ch)
        {
        case gpuf::id_cast(RGBA::R):
            d.red = s;
            break;
        case gpuf::id_cast(RGBA::G):
            d.green = s;
            break;
        case gpuf::id_cast(RGBA::B):
            d.blue = s;
            break;
        case gpuf::id_cast(RGBA::A):
            d.alpha = s;
            break;
        default:
            return;
        }
    }


    GPU_KERNAL
    static void rgb_u16_to_rgb_u8(DeviceViewRGBu16 src, DeviceView dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 3);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto s = gpuf::channel_u16_to_u8(*gpuf::channel_xy_at(src, cxy));
        auto& d = gpuf::xy_at(dst, cxy.x, cxy.y)->rgba;

        switch(cxy.ch)
        {
        case gpuf::id_cast(RGB::R):
            d.red = s;
            break;
        case gpuf::id_cast(RGB::G):
            d.green = s;
            break;
        case gpuf::id_cast(RGB::B):
            d.blue = s;
            break;
        default:
            return;
        }
    }
}


namespace simage
{
    void map_rgba(DeviceView const& src, DeviceViewRGBAu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 4;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgba_u8_to_rgba_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgba_u8_to_rgba_u16");
		assert(result);
    }


	void map_rgba(DeviceViewRGBAu16 const& src, DeviceView const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 4;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgba_u16_to_rgba_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgba_u16_to_rgba_u8");
		assert(result);
    }


    void map_rgb(DeviceView const& src, DeviceViewRGBu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 3;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u8_to_rgb_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u8_to_rgb_u16");
		assert(result);
    }


	void map_rgb(DeviceViewRGBu16 const& src, DeviceView const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 3;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u16_to_rgb_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u16_to_rgb_u8");
		assert(result);
    }
}


/* map gray */

namespace gpu
{
    GPU_KERNAL
    static void gray_u8_to_gray_u16(DeviceViewGray src, DeviceView1u16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto s = *gpuf::xy_at(src, xy);
        auto& d = *gpuf::xy_at(dst, xy);

        d = gpuf::channel_u8_to_u16(s);
    }


    GPU_KERNAL
    static void gray_u16_to_gray_u8(DeviceView1u16 src, DeviceViewGray dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto s = *gpuf::xy_at(src, xy);
        auto& d = *gpuf::xy_at(dst, xy);

        d = gpuf::channel_u16_to_u8(s);
    }
}


namespace simage
{
    void map_gray(DeviceViewGray const& src, DeviceView1u16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::gray_u8_to_gray_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::gray_u8_to_gray_u16");
		assert(result);
    }


    void map_gray(DeviceView1u16 const& src, DeviceViewGray const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::gray_u16_to_gray_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::gray_u16_to_gray_u8");
		assert(result);
    }    
}


/* map rgb gray */

namespace gpu
{
    GPU_KERNAL
    static void rgba_u8_to_gray_u8(DeviceView src, DeviceViewGray dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto s = gpuf::xy_at(src, xy)->rgba;
        auto& d = *gpuf::xy_at(dst, xy);

        d = gpuf::round_to_u8(gpuf::to_grayscale_standard(s.red, s.green, s.blue));
    }


    GPU_KERNAL
    static void rgba_u8_to_gray_u16(DeviceView src, DeviceView1u16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto s = gpuf::xy_at(src, xy)->rgba;
        auto& d = *gpuf::xy_at(dst, xy);

        d = gpuf::channel_u8_to_u16(gpuf::to_grayscale_standard(s.red, s.green, s.blue));
    }


    GPU_KERNAL
    static void rgb_u16_to_gray_u16(DeviceViewRGBu16 src, DeviceView1u16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto r = *gpuf::channel_xy_at(src, xy.x, xy.y, RGB::R);
        auto g = *gpuf::channel_xy_at(src, xy.x, xy.y, RGB::G);
        auto b = *gpuf::channel_xy_at(src, xy.x, xy.y, RGB::B);

        auto& d = *gpuf::xy_at(dst, xy);

        d = gpuf::round_to_u16(gpuf::to_grayscale_standard(r, g, b));
    }


    GPU_KERNAL
    static void gray_u16_to_rgba_u8(DeviceView1u16 src, DeviceView dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto s = *gpuf::xy_at(src, xy);
        auto& d = gpuf::xy_at(dst, xy)->rgba;

        d = {
            gpuf::channel_u16_to_u8(s),
            gpuf::channel_u16_to_u8(s),
            gpuf::channel_u16_to_u8(s),
            255
        };
    }

}


namespace simage
{
    void map_rgb_gray(DeviceView const& src, DeviceViewGray const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgba_u8_to_gray_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgba_u8_to_gray_u8");
		assert(result);
    }


    void map_rgb_gray(DeviceView const& src, DeviceView1u16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgba_u8_to_gray_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgba_u8_to_gray_u16");
		assert(result);
    }


    void map_rgb_gray(DeviceViewRGBu16 const& src, DeviceView1u16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u16_to_gray_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u16_to_gray_u16");
		assert(result);
    }


    void map_gray_rgb(DeviceView1u16 const& src, DeviceView const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::gray_u16_to_rgba_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::gray_u16_to_rgba_u8");
		assert(result);
    }
}


/* map hsv */

namespace gpu
{
    GPU_KERNAL
    static void rgb_u8_to_hsv_u16(DeviceView src, DeviceViewHSVu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto s = gpuf::xy_at(src, xy)->rgba;

        auto hsv = gpuf::rgb_u8_to_hsv_u16(s.red, s.green, s.blue);

        *gpuf::channel_xy_at(dst, xy.x, xy.y, HSV::H) = hsv.hue;
        *gpuf::channel_xy_at(dst, xy.x, xy.y, HSV::S) = hsv.sat;
        *gpuf::channel_xy_at(dst, xy.x, xy.y, HSV::V) = hsv.val;
    }


    GPU_KERNAL
    static void rgb_u16_to_hsv_u16(DeviceViewRGBu16 src, DeviceViewHSVu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto r = *gpuf::channel_xy_at(src, xy.x, xy.y, RGB::R);
        auto g = *gpuf::channel_xy_at(src, xy.x, xy.y, RGB::G);
        auto b = *gpuf::channel_xy_at(src, xy.x, xy.y, RGB::B);

        auto hsv = gpuf::rgb_u16_to_hsv_u16(r, g, b);

        *gpuf::channel_xy_at(dst, xy.x, xy.y, HSV::H) = hsv.hue;
        *gpuf::channel_xy_at(dst, xy.x, xy.y, HSV::S) = hsv.sat;
        *gpuf::channel_xy_at(dst, xy.x, xy.y, HSV::V) = hsv.val;
    }


    GPU_KERNAL
    static void hsv_u16_to_rgba_u8(DeviceViewHSVu16 src, DeviceView dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto h = *gpuf::channel_xy_at(src, xy.x, xy.y, HSV::H);
        auto s = *gpuf::channel_xy_at(src, xy.x, xy.y, HSV::S);
        auto v = *gpuf::channel_xy_at(src, xy.x, xy.y, HSV::V);

        auto& d = gpuf::xy_at(dst, xy.x, xy.y)->rgba;

        /*if (s == 0 || v == 0)
        {
            d = { 0, 0, 0, 255 };
        }
        else 
        {
            d = gpuf::hsv_u16_to_rgba_u8(h, CH_U16_MAX, CH_U16_MAX);
        }*/

        d = gpuf::hsv_u16_to_rgba_u8(h, s, v);
    }


    GPU_KERNAL
    static void hsv_u16_to_rgb_u16(DeviceViewHSVu16 src, DeviceViewRGBu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height);

        auto xy = gpuf::get_thread_xy(src, t);

        auto h = *gpuf::channel_xy_at(src, xy.x, xy.y, HSV::H);
        auto s = *gpuf::channel_xy_at(src, xy.x, xy.y, HSV::S);
        auto v = *gpuf::channel_xy_at(src, xy.x, xy.y, HSV::V);

        auto rgb = gpuf::hsv_u16_to_rgb_u16(h, s, v);

        *gpuf::channel_xy_at(dst, xy.x, xy.y, RGB::R) = rgb.red;
        *gpuf::channel_xy_at(dst, xy.x, xy.y, RGB::G) = rgb.green;
        *gpuf::channel_xy_at(dst, xy.x, xy.y, RGB::B) = rgb.blue;
    }
}


namespace simage
{
    void map_rgb_hsv(DeviceView const& src, DeviceViewHSVu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u8_to_hsv_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u8_to_hsv_u16");
		assert(result);
    }


    void map_rgb_hsv(DeviceViewRGBu16 const& src, DeviceViewHSVu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u16_to_hsv_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u16_to_hsv_u16");
		assert(result);
    }


    void map_hsv_rgb(DeviceViewHSVu16 const& src, DeviceView const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::hsv_u16_to_rgba_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::hsv_u16_to_rgba_u8");
		assert(result);
    }


    void map_hsv_rgb(DeviceViewHSVu16 const& src, DeviceViewRGBu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::hsv_u16_to_rgb_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::hsv_u16_to_rgb_u16");
		assert(result);
    }
}


/* map yuv */

namespace gpu
{
    GPU_KERNAL
    static void rgb_u8_to_yuv_u16(DeviceView src, DeviceViewYUVu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 3);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto rgba = gpuf::xy_at(src, cxy.x, cxy.y)->rgba;
        auto r = gpuf::channel_u8_to_r32(rgba.red);
        auto g = gpuf::channel_u8_to_r32(rgba.green);
        auto b = gpuf::channel_u8_to_r32(rgba.blue);
        
        r32 value = 1.0f;

        auto& d = *gpuf::channel_xy_at(dst, cxy);

        switch(cxy.ch)
        {
        case gpuf::id_cast(YUV::Y):
            value = gpuf::rgb_to_yuv_y(r, g, b);
            break;
        case gpuf::id_cast(YUV::U):
            value = gpuf::rgb_to_yuv_u(r, g, b);
            break;
        case gpuf::id_cast(YUV::V):
            value = gpuf::rgb_to_yuv_v(r, g, b);
            break;
        default:
            return;
        }

        d = gpuf::channel_r32_to_u16(value);
    }


    GPU_KERNAL
    static void rgb_u16_to_yuv_u16(DeviceViewRGBu16 src, DeviceViewYUVu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 3);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto r16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, RGB::R);
        auto g16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, RGB::G);
        auto b16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, RGB::B);

        auto r = gpuf::channel_u16_to_r32(r16);
        auto g = gpuf::channel_u16_to_r32(g16);
        auto b = gpuf::channel_u16_to_r32(b16);
        
        r32 value = 1.0f;

        auto& d = *gpuf::channel_xy_at(dst, cxy);

        switch(cxy.ch)
        {
        case gpuf::id_cast(YUV::Y):
            value = gpuf::rgb_to_yuv_y(r, g, b);
            break;
        case gpuf::id_cast(YUV::U):
            value = gpuf::rgb_to_yuv_u(r, g, b);
            break;
        case gpuf::id_cast(YUV::V):
            value = gpuf::rgb_to_yuv_v(r, g, b);
            break;
        default:
            return;
        }

        d = gpuf::channel_r32_to_u16(value);
    }


    GPU_KERNAL
    static void yuv_u16_to_rgb_u8(DeviceViewYUVu16 src, DeviceView dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 3);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto y16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, YUV::Y);
        auto u16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, YUV::U);
        auto v16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, YUV::V);

        auto y = gpuf::channel_u16_to_r32(y16);
        auto u = gpuf::channel_u16_to_r32(u16);
        auto v = gpuf::channel_u16_to_r32(v16);

        r32 value = 1.0f;

        auto& d = gpuf::xy_at(dst, cxy.x, cxy.y)->channels[cxy.ch];

        switch(cxy.ch)
        {
        case gpuf::id_cast(RGB::R):
            value = gpuf::yuv_to_rgb_r(y, u, v);
            break;
        case gpuf::id_cast(RGB::G):
            value = gpuf::yuv_to_rgb_g(y, u, v);
            break;
        case gpuf::id_cast(RGB::B):
            value = gpuf::yuv_to_rgb_b(y, u, v);
            break;
        default:
            return;
        }

        d = gpuf::channel_r32_to_u8(value);
    }


    GPU_KERNAL
    static void yuv_u16_to_rgb_u16(DeviceViewYUVu16 src, DeviceViewRGBu16 dst, u32 n_threads)
    {
        auto t = blockDim.x * blockIdx.x + threadIdx.x;
		if (t >= n_threads)
		{
			return;
		}

        assert(n_threads == src.width * src.height * 3);

        auto cxy = gpuf::get_thread_channel_xy(src, t);

        auto y16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, YUV::Y);
        auto u16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, YUV::U);
        auto v16 = *gpuf::channel_xy_at(src, cxy.x, cxy.y, YUV::V);

        auto y = gpuf::channel_u16_to_r32(y16);
        auto u = gpuf::channel_u16_to_r32(u16);
        auto v = gpuf::channel_u16_to_r32(v16);

        r32 value = 1.0f;

        auto& d = *gpuf::channel_xy_at(dst, cxy.x, cxy.y, cxy.ch);

        switch(cxy.ch)
        {
        case gpuf::id_cast(RGB::R):
            value = gpuf::yuv_to_rgb_r(y, u, v);
            break;
        case gpuf::id_cast(RGB::G):
            value = gpuf::yuv_to_rgb_g(y, u, v);
            break;
        case gpuf::id_cast(RGB::B):
            value = gpuf::yuv_to_rgb_b(y, u, v);
            break;
        default:
            return;
        }

        d = gpuf::channel_r32_to_u16(value);
    }
}


/* map yuv */

namespace simage
{
	void map_rgb_yuv(DeviceView const& src, DeviceViewYUVu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 3;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u8_to_yuv_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u8_to_yuv_u16");
		assert(result);
    }


	void map_rgb_yuv(DeviceViewRGBu16 const& src, DeviceViewYUVu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 3;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::rgb_u16_to_yuv_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::rgb_u16_to_yuv_u16");
		assert(result);
    }


	void map_yuv_rgb(DeviceViewYUVu16 const& src, DeviceView const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 3;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::yuv_u16_to_rgb_u8, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::yuv_u16_to_rgb_u8");
		assert(result);
    }


	void map_yuv_rgb(DeviceViewYUVu16 const& src, DeviceViewRGBu16 const& dst)
    {
        assert(verify(src, dst));

        auto const width = src.width;
		auto const height = src.height;

		auto const n_threads = width * height * 3;
		auto const n_blocks = calc_thread_blocks(n_threads);
		constexpr auto block_size = THREADS_PER_BLOCK;

        cuda_launch_kernel(gpu::yuv_u16_to_rgb_u16, n_blocks, block_size, src, dst, n_threads);

        auto result = cuda::launch_success("gpu::yuv_u16_to_rgb_u16");
		assert(result);
    }
}