#include "device.hpp"
#include "cuda_def.cuh"

#include <cassert>
#include <cstddef>

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cstring>

#endif


static void check_error(hipError_t err, cstr label = "")
{
    if (err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR
    #ifndef	NDEBUG

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));

    if (std::strlen(label))
    {
        printf("\n%s", label);
    }
    
    printf("\n\n******************\n\n");

    #endif
    #endif
}


namespace cuda
{
    u8* device_malloc(size_t n_bytes)
    {
        assert(n_bytes);

        u8* data;

        auto err = hipMalloc((void**)&(data), n_bytes);
        check_error(err, "malloc");

        if (err != hipSuccess)
        {
            return nullptr;
        }

        return data;
    }


    u8* unified_malloc(size_t n_bytes)
    {
        assert(n_bytes);

        u8* data;

        auto err = hipMallocManaged((void**)&(data), n_bytes);
        check_error(err, "malloc");

        if (err != hipSuccess)
        {
            return nullptr;
        }

        return data;
    }


    bool free(void* data)
    {
        if (data)
        {
            auto err = hipFree(data);
            check_error(err, "free");

            return err == hipSuccess;
        }

        return true;
    }
    

    bool memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
    {
        hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
        check_error(err, "memcpy_to_device");

        return err == hipSuccess;
    }


    bool memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
    {
        hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
        check_error(err, "memcpy_to_host");

        return err == hipSuccess;
    }


    bool no_errors(cstr label)
    {
        #ifndef	NDEBUG

        hipError_t err = hipGetLastError();
        check_error(err, label);

        return err == hipSuccess;

        #else

        return true;

        #endif
    }


    bool launch_success(cstr label)
    {
        #ifndef	NDEBUG

        hipError_t err = hipDeviceSynchronize();
        check_error(err, label);

        return err == hipSuccess;

        #else

        return true;

        #endif
    }
}